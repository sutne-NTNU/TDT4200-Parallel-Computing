#include "hip/hip_runtime.h"
#include <iostream>
#include <sstream>

#define STB_IMAGE_IMPLEMENTATION
#include <stb/stb_image.h>

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include <stb/stb_image_write.h>

using namespace std;

typedef struct pix
{
    unsigned char r, g, b, a;
} pixel;

typedef struct SimplePoint_struct
{
    float x, y;
} SimplePoint;

typedef struct SimpleFeatureLine_struct
{
    SimplePoint startPoint;
    SimplePoint endPoint;
} SimpleFeatureLine;

template <typename T>
__host__ __device__ T CLAMP(T value, T low, T high)
{
    return (value < low) ? low : ((value > high) ? high : value);
}

#define cudaErrorCheck(ans)                   \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code == hipSuccess) return;
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
}

//////////////////////////////////////////////////////////
// GLOBALS                                              //
int imageWidth, imageHeight, numLines, steps;           //
float p, a, b, t, stepSize;                             //
pixel *sourceImage, *destinationImage;                  //
SimpleFeatureLine *sourceLines, *destinationLines;      //
string outputPath;                                      //
//////////////////////////////////////////////////////////

void imgRead(string filename, pixel *&map, int &imgW, int &imgH)
{
    if (filename.empty())
    {
        cout << "The input file name cannot be empty" << endl;
        exit(1);
    }
    stbi_set_flip_vertically_on_load(true);
    int x, y, componentsPerPixel;
    map = (pixel *)stbi_load(filename.c_str(), &x, &y, &componentsPerPixel, STBI_rgb_alpha);
    
    if(imgW != 0 || imgH != 0)
    {
        if(x != imgW || y != imgH)
        {
            fprintf(stderr, "Images must have equal dimensions! but was: %dx%d and %dx%d\n", imgW, imgH, x, y);
            exit(1);
        }
    }
    else
    {
        imgW = x;
        imgH = y;
    }
    cout << "Loaded image from: \t\"" << filename << "\"" << endl;
}

void imgWrite(string filename, pixel *map, int imgW, int imgH)
{
    if (filename.empty())
    {
        cout << "The output file name cannot be empty" << endl;
        exit(1);
    }
    stbi_flip_vertically_on_write(true);
    stbi_write_png(filename.c_str(), imgW, imgH, STBI_rgb_alpha, map, sizeof(pixel) * imgW);
}

void loadLines(const char* filename, SimpleFeatureLine *&linesSrc, SimpleFeatureLine *&linesDst, int *numLines)
{
    FILE *f = fopen(filename, "r");
    if (f == NULL)
    {
        printf("Error opening file %s! \n", filename);
        exit(1);
    }
    fscanf(f, "%d", numLines);
    linesSrc = (SimpleFeatureLine *)malloc(sizeof(SimpleFeatureLine) * (*numLines));
    linesDst = (SimpleFeatureLine *)malloc(sizeof(SimpleFeatureLine) * (*numLines));
    SimpleFeatureLine *line;
    int fac = 2;
    for (int i = 0; i < (*numLines) * fac; i++)
    {
        line = (i % fac) ? &linesDst[(i - 1) / fac] : &linesSrc[i / fac];
        fscanf(f, "%f,%f,%f,%f", &(line->startPoint.x), &(line->startPoint.y),
               &(line->endPoint.x), &(line->endPoint.y));
    }
    printf("Loaded %d lines from: \t\"%s\"\n", *numLines, filename);
}

void simpleLineInterpolate(SimpleFeatureLine *sourceLines,
                           SimpleFeatureLine *destLines,
                           SimpleFeatureLine **allMorphLines,
                           int numLines, float t)
{
    SimpleFeatureLine *interLines = (SimpleFeatureLine *)malloc(sizeof(SimpleFeatureLine) * numLines);
    for (int i = 0; i < numLines; i++)
    {
        interLines[i].startPoint.x = (1 - t) * (sourceLines[i].startPoint.x) //
                                     + t * (destLines[i].startPoint.x);      //
        interLines[i].startPoint.y = (1 - t) * (sourceLines[i].startPoint.y) //
                                     + t * (destLines[i].startPoint.y);      //
        interLines[i].endPoint.x = (1 - t) * (sourceLines[i].endPoint.x)     //
                                   + t * (destLines[i].endPoint.x);          //
        interLines[i].endPoint.y = (1 - t) * (sourceLines[i].endPoint.y)     //
                                   + t * (destLines[i].endPoint.y);          //
    }
    *allMorphLines = interLines;
}

__host__ __device__ void warp(const SimplePoint *interPt,
                              SimpleFeatureLine *interLines,
                              SimpleFeatureLine *sourceLines,
                              const int sourceLinesSize,
                              SimplePoint *src)
{
    int i;
    float interLength, srcLength;
    float weight, weightSum, dist;
    float sum_x, sum_y; // weighted sum of the coordination of the point "src"
    float u, v;
    SimplePoint pd, pq, qd;
    float X, Y;

    sum_x = 0;
    sum_y = 0;
    weightSum = 0;

    for (i = 0; i < sourceLinesSize; i++)
    {
        pd.x = interPt->x - interLines[i].startPoint.x;
        pd.y = interPt->y - interLines[i].startPoint.y;
        pq.x = interLines[i].endPoint.x - interLines[i].startPoint.x;
        pq.y = interLines[i].endPoint.y - interLines[i].startPoint.y;
        interLength = pq.x * pq.x + pq.y * pq.y;
        u = (pd.x * pq.x + pd.y * pq.y) / interLength;

        interLength = sqrt(interLength); // length of the vector PQ

        v = (pd.x * pq.y - pd.y * pq.x) / interLength;

        pq.x = sourceLines[i].endPoint.x - sourceLines[i].startPoint.x;
        pq.y = sourceLines[i].endPoint.y - sourceLines[i].startPoint.y;

        srcLength = sqrt(pq.x * pq.x + pq.y * pq.y); // length of the vector P'Q'
        // corresponding point based on the ith line
        X = sourceLines[i].startPoint.x + u * pq.x + v * pq.y / srcLength;
        Y = sourceLines[i].startPoint.y + u * pq.y - v * pq.x / srcLength;

        // the distance from the corresponding point to the line P'Q'
        if (u < 0)
            dist = sqrt(pd.x * pd.x + pd.y * pd.y);
        else if (u > 1)
        {
            qd.x = interPt->x - interLines[i].endPoint.x;
            qd.y = interPt->y - interLines[i].endPoint.y;
            dist = sqrt(qd.x * qd.x + qd.y * qd.y);
        }
        else
        {
            dist = abs(v);
        }

        weight = pow(1.0f / (1.0f + dist), 2.0f);
        sum_x += X * weight;
        sum_y += Y * weight;
        weightSum += weight;
    }

    src->x = sum_x / weightSum;
    src->y = sum_y / weightSum;
}

__host__ __device__ void bilinear(pixel *Im, float row, float col, pixel *pix, int dImgWidth)
{
    int cm = (int)ceil(row);
    int fm = (int)floor(row);
    int cn = (int)ceil(col);
    int fn = (int)floor(col);
    double alpha = ceil(row) - row;
    double beta = ceil(col) - col;
    pix->r = (unsigned int)(alpha * beta * Im[fm * dImgWidth + fn].r                 //
                            + (1 - alpha) * beta * Im[cm * dImgWidth + fn].r         //
                            + alpha * (1 - beta) * Im[fm * dImgWidth + cn].r         //
                            + (1 - alpha) * (1 - beta) * Im[cm * dImgWidth + cn].r); //
    pix->g = (unsigned int)(alpha * beta * Im[fm * dImgWidth + fn].g                 //
                            + (1 - alpha) * beta * Im[cm * dImgWidth + fn].g         //
                            + alpha * (1 - beta) * Im[fm * dImgWidth + cn].g         //
                            + (1 - alpha) * (1 - beta) * Im[cm * dImgWidth + cn].g); //
    pix->b = (unsigned int)(alpha * beta * Im[fm * dImgWidth + fn].b                 //
                            + (1 - alpha) * beta * Im[cm * dImgWidth + fn].b         //
                            + alpha * (1 - beta) * Im[fm * dImgWidth + cn].b         //
                            + (1 - alpha) * (1 - beta) * Im[cm * dImgWidth + cn].b); //
    pix->a = 255;
}

__host__ __device__ void ColorInterPolate(const SimplePoint *Src_P,
                                          const SimplePoint *Dest_P, float t,
                                          pixel *imgSrc, pixel *imgDest,
                                          pixel *rgb, int dImgWidth)
{
    pixel srcColor, destColor;

    bilinear(imgSrc, Src_P->y, Src_P->x, &srcColor, dImgWidth);
    bilinear(imgDest, Dest_P->y, Dest_P->x, &destColor, dImgWidth);

    rgb->b = srcColor.b * (1 - t) + destColor.b * t;
    rgb->g = srcColor.g * (1 - t) + destColor.g * t;
    rgb->r = srcColor.r * (1 - t) + destColor.r * t;
    rgb->a = 255;
}

///////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////

/** Parses all arguments and reads the input images and lines */
void parseAndReadFiles(int argc, char *argv[])
{
    if (!(argc == 6 || argc == 9)) // has to be either 6 or 9
    {
        cout << "Usage: ./morph srcImg.png destImg.png lines.txt outputPath steps [p] [a] [b]" << endl;
        exit(1);
    }
    string fileSourceImage = argv[1];
    string fileDestinationImage = argv[2];
    string fileLines = argv[3];
    outputPath = argv[4];
    istringstream(argv[5]) >> steps;
    stepSize = 1.0 / steps;
    if (argc == 9)
    {
        istringstream(argv[6]) >> p;
        istringstream(argv[7]) >> a;
        istringstream(argv[8]) >> b;
    }
    else
    {
        p = 0;
        a = 1;
        b = 2;
    }
    t = 0.5;
    imgRead(fileSourceImage, sourceImage, imageWidth, imageHeight);
    imgRead(fileDestinationImage, destinationImage, imageWidth, imageHeight);
    loadLines(fileLines.c_str(), sourceLines, destinationLines, &numLines);
}

/** Using the total steps and the currently completed step to print a progressbar.
 *  The progress should overwrite itself until it reaches 100%.*/
 void printProgress(const char* prefix, int step, int total)
 {
     const int increments = 50;
     double percent_completed = 100.0 * (double)(step + 1.0) / total;
     printf("\r%s: %.0f%% |", prefix, percent_completed);
     for (int i = 0; i < increments; ++i)
     {
         double bar_percent = 100.0 * (double)i / increments;
         printf(bar_percent <= percent_completed ? "█" : " ");
     }
     printf("| %d/%d", step + 1, total);
     fflush(stdout);
     if (step + 1 == total) printf("\n");
 }

/** Start measuring CUDA time */
void cuda_time_start(hipEvent_t *start, hipEvent_t *stop)
{
    cudaErrorCheck(hipEventCreate(start));
    cudaErrorCheck(hipEventCreate(stop));
    cudaErrorCheck(hipEventRecord(*start, 0));
}

/** Stop measuring cuda time and return elapsed time in ms */
float cuda_time_stop(hipEvent_t *start, hipEvent_t *stop, bool synchronize)
{
    float elapsed = 0;
    if (synchronize) cudaErrorCheck(hipDeviceSynchronize());
    cudaErrorCheck(hipEventRecord(*stop, 0));
    cudaErrorCheck(hipEventSynchronize(*stop));
    cudaErrorCheck(hipEventElapsedTime(&elapsed, *start, *stop));
    cudaErrorCheck(hipEventDestroy(*start));
    cudaErrorCheck(hipEventDestroy(*stop));
    return elapsed;
}

__global__ void morphKernel(SimpleFeatureLine *sourceLines,
                            SimpleFeatureLine *destinationLines,
                            SimpleFeatureLine *morphLines, 
                            pixel *sourceImage, 
                            pixel *destinationImage, 
                            pixel *morphedImage, 
                            int imageWidth, int imageHeight, 
                            int numLines, float dT)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    if (imageWidth <= x || imageHeight <= y) return; // Out of range of image
    
    extern __shared__ SimpleFeatureLine lines[]; // Shared memory for all feature lines
    // Split shared memory into three sections for the three feature lines
    SimpleFeatureLine *sSrcLines = &lines[0 * numLines];
    SimpleFeatureLine *sDstLines = &lines[1 * numLines];
    SimpleFeatureLine *sMrpLines = &lines[2 * numLines];

    // Each thread fills a specific index from the global to shared memory for its block
    // This assumes that there are more pixels/threads in each block than lines, which i would 
    // say is a fair assumption (in Part 2 number of threads per block is 64, and numLines is 33)
    int blockIndex = threadIdx.y * blockDim.x + threadIdx.x;
    if (blockIndex < numLines) 
    {
        sSrcLines[blockIndex] = sourceLines[blockIndex];
        sDstLines[blockIndex] = destinationLines[blockIndex];
        sMrpLines[blockIndex] = morphLines[blockIndex];
    }
    __syncthreads(); // wait for all shared memory to be ready in this block

    SimplePoint q{.x = float(x), .y = float(y)};
    SimplePoint src, dest;

    warp(&q, sMrpLines, sSrcLines, numLines, &src);
    warp(&q, sMrpLines, sDstLines, numLines, &dest);

    src.x = CLAMP<double>(src.x, 0, imageWidth - 1);
    src.y = CLAMP<double>(src.y, 0, imageHeight - 1);
    dest.x = CLAMP<double>(dest.x, 0, imageWidth - 1);
    dest.y = CLAMP<double>(dest.y, 0, imageHeight - 1);

    pixel interColor;
    ColorInterPolate(&src, &dest, dT, sourceImage, destinationImage, &interColor, imageWidth);

    int index = y * imageWidth + x;
    morphedImage[index].r = interColor.r;
    morphedImage[index].g = interColor.g;
    morphedImage[index].b = interColor.b;
    morphedImage[index].a = interColor.a;
}




int main(int argc, char *argv[])
{
    parseAndReadFiles(argc, argv);

    // Calculate all sizes
    size_t morphArrSize = sizeof(pixel *) * (steps + 1);
    size_t lineArrSize = sizeof(SimpleFeatureLine *) * (steps + 1);
    size_t imageSize = sizeof(pixel) * imageWidth * imageHeight;
    size_t lineSize = sizeof(SimpleFeatureLine) * numLines;
    // Shared memory will contain sourceLines, destinationLines and morphLines 
    size_t sharedMemSize = 3 * lineSize;

    // Create arrays for all outputimages and all the morph lines
    pixel **morphedImages = (pixel **)malloc(morphArrSize);
    SimpleFeatureLine **allMorphLines = (SimpleFeatureLine **)malloc(lineArrSize);
    for (int i = 0; i < steps + 1; i++)
    {
        morphedImages[i] = (pixel *)malloc(imageSize);
        simpleLineInterpolate(sourceLines, destinationLines, &(allMorphLines[i]), numLines, t);
    }

    // Start total time measuring
    hipEvent_t start_total, stop_total;
    cuda_time_start(&start_total, &stop_total);

    // Allocate space on device (GPU) for lines and images
    pixel *dSourceImage, *dDestinationImage, *dMorphedImage;
    hipMalloc((void **)&dSourceImage, imageSize);
    hipMalloc((void **)&dDestinationImage, imageSize);
    hipMalloc((void **)&dMorphedImage, imageSize);
    SimpleFeatureLine *dSourceLines, *dDestinationLines, *dMorphLines;
    hipMalloc((void **)&dSourceLines, lineSize);
    hipMalloc((void **)&dDestinationLines, lineSize);
    hipMalloc((void **)&dMorphLines, lineSize);

    // Copy source and destination data to device
    hipMemcpy(dSourceImage, sourceImage, imageSize, hipMemcpyHostToDevice);
    hipMemcpy(dSourceLines, sourceLines, lineSize, hipMemcpyHostToDevice);
    hipMemcpy(dDestinationImage, destinationImage, imageSize, hipMemcpyHostToDevice);
    hipMemcpy(dDestinationLines, destinationLines, lineSize, hipMemcpyHostToDevice);

    // Estimate optimal blockSize 
    int optimTotalBlockSize, minGridSize;
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &optimTotalBlockSize, morphKernel, sharedMemSize, 0); 
    int optimBlockSize = sqrt(optimTotalBlockSize);

    // Defining Block and Grid Size
    dim3 blockSize(optimBlockSize, optimBlockSize);
    int num_blocks_x = (imageWidth / blockSize.x);
    int num_blocks_y = (imageHeight / blockSize.y);
    dim3 gridSize(num_blocks_x, num_blocks_y);
    printf("Using: \tblockSize = %dx%d\n\tgridSize = %dx%d\n", blockSize.x, blockSize.y, gridSize.x, gridSize.y);

    for (int i = 0; i < steps + 1; i++)
    {
        // Copy morph lines for this step to device
        hipMemcpy(dMorphLines, allMorphLines[i], lineSize, hipMemcpyHostToDevice);

        hipEvent_t start, stop;
        cuda_time_start(&start, &stop);
        // Launching Kernel
        morphKernel<<<gridSize, blockSize, sharedMemSize>>>(
            dSourceLines, dDestinationLines, dMorphLines,  
            dSourceImage, dDestinationImage, dMorphedImage,  
            imageWidth, imageHeight, numLines, i * stepSize
        );
        float time_this_step = cuda_time_stop(&start, &stop, false);
        printf("Time in morphKernel (step %d): %.2f ms\n", i, time_this_step);

        // Copy morphed image for this step from device to host
        hipMemcpy(morphedImages[i], dMorphedImage, imageSize, hipMemcpyDeviceToHost);
    }
    float time_total = cuda_time_stop(&start_total, &stop_total, true);
    printf("Total time in GPU: %.2f ms\n", time_total);

    // Free all Cuda memory
    hipFree(dSourceImage);
    hipFree(dSourceLines);
    hipFree(dDestinationImage);
    hipFree(dDestinationLines);
    hipFree(dMorphedImage);
    hipFree(dMorphLines);

    // Write the morphed images to file and free the host memory
    for (int i = 0; i < steps + 1; i++)
    {
        imgWrite(outputPath + to_string(stepSize * i) + ".png", morphedImages[i], imageWidth, imageHeight);
        printProgress("Writing Images To File", i, steps + 1);
        free(morphedImages[i]);
        free(allMorphLines[i]);
    }
    free(morphedImages);
    free(allMorphLines);
    return 0;
}
