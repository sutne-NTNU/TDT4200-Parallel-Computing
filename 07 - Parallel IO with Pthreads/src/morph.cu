#include "hip/hip_runtime.h"
#include <iostream>
#include <sstream>
#include <sys/time.h>
#include <pthread.h>

#define STB_IMAGE_IMPLEMENTATION
#include <stb/stb_image.h>

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include <stb/stb_image_write.h>

#define WALLTIME(t) ((double)(t).tv_sec + 1e-6 * (double)(t).tv_usec)

using namespace std;

typedef struct pix
{
    unsigned char r, g, b, a;
} pixel;

typedef struct SimplePoint_struct
{
    float x, y;
} SimplePoint;

typedef struct SimpleFeatureLine_struct
{
    SimplePoint startPoint;
    SimplePoint endPoint;
} SimpleFeatureLine;


template <typename T>
__host__ __device__ T CLAMP(T value, T low, T high)
{
    return (value < low) ? low : ((value > high) ? high : value);
}

#define cudaErrorCheck(ans)                   \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code == hipSuccess) return;
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
}

//////////////////////////////////////////////////////////
// GLOBALS                                              //
int imageWidth, imageHeight, numLines, steps;           //
float p, a, b, t, stepSize;                             //
pixel *sourceImage, *destinationImage;                  //
SimpleFeatureLine *sourceLines, *destinationLines;      //
string outputPath;                                      //
pixel **morphedImages;                                  //
SimpleFeatureLine **allMorphLines;                      //
//////////////////////////////////////////////////////////

/** Using the total steps and the currently completed step to print a progressbar.
*  The progress overwrites itself until it reaches 100%.*/
void printProgress(const char *prefix, int step, int total)
{
    const int increments = 50;
    double percent_completed = 100.0 * (double)step / total;
    printf("\r%s: \t%.0f%%\t|", prefix, percent_completed);
    for (int i = 0; i < increments; ++i)
    {
        double bar_percent = 100.0 * (double)i / increments;
        printf(bar_percent < percent_completed ? "█" : " ");
    }
    printf("| %d/%d", step, total);
    fflush(stdout);
    if (step == total) printf("\n");
}

void imgRead(string filename, pixel *&map, int &imgW, int &imgH)
{
    if (filename.empty())
    {
        cout << "The input file name cannot be empty" << endl;
        exit(1);
    }
    stbi_set_flip_vertically_on_load(true);
    int x, y, componentsPerPixel;
    map = (pixel *)stbi_load(filename.c_str(), &x, &y, &componentsPerPixel, STBI_rgb_alpha);

    if (imgW != 0 || imgH != 0)
    {
        if (x != imgW || y != imgH)
        {
            fprintf(stderr, "Images must have equal dimensions! but was: %dx%d and %dx%d\n", imgW, imgH, x, y);
            exit(1);
        }
    }
    else
    {
        imgW = x;
        imgH = y;
    }
    cout << "Loaded image from: \t\"" << filename << "\"" << endl;
}

void imgWrite(string filename, pixel *map, int imgW, int imgH)
{
    if (filename.empty())
    {
        cout << "The output file name cannot be empty" << endl;
        exit(1);
    }
    stbi_flip_vertically_on_write(true);
    stbi_write_png(filename.c_str(), imgW, imgH, STBI_rgb_alpha, map, sizeof(pixel) * imgW);
}

void loadLines(const char *filename, SimpleFeatureLine *&linesSrc, SimpleFeatureLine *&linesDst, int *numLines)
{
    FILE *f = fopen(filename, "r");
    if (f == NULL)
    {
        printf("Error opening file %s! \n", filename);
        exit(1);
    }
    fscanf(f, "%d", numLines);
    linesSrc = (SimpleFeatureLine *)malloc(sizeof(SimpleFeatureLine) * (*numLines));
    linesDst = (SimpleFeatureLine *)malloc(sizeof(SimpleFeatureLine) * (*numLines));
    SimpleFeatureLine *line;
    int fac = 2;
    for (int i = 0; i < (*numLines) * fac; i++)
    {
        line = (i % fac) ? &linesDst[(i - 1) / fac] : &linesSrc[i / fac];
        fscanf(f, "%f,%f,%f,%f", &(line->startPoint.x), &(line->startPoint.y),
               &(line->endPoint.x), &(line->endPoint.y));
    }
    printf("Loaded %d lines from: \t\"%s\"\n", *numLines, filename);
}

void simpleLineInterpolate(SimpleFeatureLine *sourceLines,
                           SimpleFeatureLine *destLines,
                           SimpleFeatureLine **allMorphLines,
                           int numLines, float t)
{
    SimpleFeatureLine *interLines = (SimpleFeatureLine *)malloc(sizeof(SimpleFeatureLine) * numLines);
    for (int i = 0; i < numLines; i++)
    {
        interLines[i].startPoint.x = (1 - t) * (sourceLines[i].startPoint.x) //
                                     + t * (destLines[i].startPoint.x);      //
        interLines[i].startPoint.y = (1 - t) * (sourceLines[i].startPoint.y) //
                                     + t * (destLines[i].startPoint.y);      //
        interLines[i].endPoint.x = (1 - t) * (sourceLines[i].endPoint.x)     //
                                   + t * (destLines[i].endPoint.x);          //
        interLines[i].endPoint.y = (1 - t) * (sourceLines[i].endPoint.y)     //
                                   + t * (destLines[i].endPoint.y);          //
    }
    *allMorphLines = interLines;
}

__host__ __device__ void warp(const SimplePoint *interPt,
                              SimpleFeatureLine *interLines,
                              SimpleFeatureLine *sourceLines,
                              const int sourceLinesSize,
                              SimplePoint *src)
{
    int i;
    float interLength, srcLength;
    float weight, weightSum, dist;
    float sum_x, sum_y; // weighted sum of the coordination of the point "src"
    float u, v;
    SimplePoint pd, pq, qd;
    float X, Y;

    sum_x = 0;
    sum_y = 0;
    weightSum = 0;

    for (i = 0; i < sourceLinesSize; i++)
    {
        pd.x = interPt->x - interLines[i].startPoint.x;
        pd.y = interPt->y - interLines[i].startPoint.y;
        pq.x = interLines[i].endPoint.x - interLines[i].startPoint.x;
        pq.y = interLines[i].endPoint.y - interLines[i].startPoint.y;
        interLength = pq.x * pq.x + pq.y * pq.y;
        u = (pd.x * pq.x + pd.y * pq.y) / interLength;

        interLength = sqrt(interLength); // length of the vector PQ

        v = (pd.x * pq.y - pd.y * pq.x) / interLength;

        pq.x = sourceLines[i].endPoint.x - sourceLines[i].startPoint.x;
        pq.y = sourceLines[i].endPoint.y - sourceLines[i].startPoint.y;

        srcLength = sqrt(pq.x * pq.x + pq.y * pq.y); // length of the vector P'Q'
        // corresponding point based on the ith line
        X = sourceLines[i].startPoint.x + u * pq.x + v * pq.y / srcLength;
        Y = sourceLines[i].startPoint.y + u * pq.y - v * pq.x / srcLength;

        // the distance from the corresponding point to the line P'Q'
        if (u < 0)
            dist = sqrt(pd.x * pd.x + pd.y * pd.y);
        else if (u > 1)
        {
            qd.x = interPt->x - interLines[i].endPoint.x;
            qd.y = interPt->y - interLines[i].endPoint.y;
            dist = sqrt(qd.x * qd.x + qd.y * qd.y);
        }
        else
        {
            dist = abs(v);
        }

        weight = pow(1.0f / (1.0f + dist), 2.0f);
        sum_x += X * weight;
        sum_y += Y * weight;
        weightSum += weight;
    }

    src->x = sum_x / weightSum;
    src->y = sum_y / weightSum;
}

__host__ __device__ void bilinear(pixel *Im, float row, float col, pixel *pix, int dImgWidth)
{
    int cm = (int)ceil(row);
    int fm = (int)floor(row);
    int cn = (int)ceil(col);
    int fn = (int)floor(col);
    double alpha = ceil(row) - row;
    double beta = ceil(col) - col;
    pix->r = (unsigned int)(alpha * beta * Im[fm * dImgWidth + fn].r                 //
                            + (1 - alpha) * beta * Im[cm * dImgWidth + fn].r         //
                            + alpha * (1 - beta) * Im[fm * dImgWidth + cn].r         //
                            + (1 - alpha) * (1 - beta) * Im[cm * dImgWidth + cn].r); //
    pix->g = (unsigned int)(alpha * beta * Im[fm * dImgWidth + fn].g                 //
                            + (1 - alpha) * beta * Im[cm * dImgWidth + fn].g         //
                            + alpha * (1 - beta) * Im[fm * dImgWidth + cn].g         //
                            + (1 - alpha) * (1 - beta) * Im[cm * dImgWidth + cn].g); //
    pix->b = (unsigned int)(alpha * beta * Im[fm * dImgWidth + fn].b                 //
                            + (1 - alpha) * beta * Im[cm * dImgWidth + fn].b         //
                            + alpha * (1 - beta) * Im[fm * dImgWidth + cn].b         //
                            + (1 - alpha) * (1 - beta) * Im[cm * dImgWidth + cn].b); //
    pix->a = 255;
}

__host__ __device__ void ColorInterPolate(const SimplePoint *Src_P,
                                          const SimplePoint *Dest_P, float t,
                                          pixel *imgSrc, pixel *imgDest,
                                          pixel *rgb, int dImgWidth)
{
    pixel srcColor, destColor;

    bilinear(imgSrc, Src_P->y, Src_P->x, &srcColor, dImgWidth);
    bilinear(imgDest, Dest_P->y, Dest_P->x, &destColor, dImgWidth);

    rgb->b = srcColor.b * (1 - t) + destColor.b * t;
    rgb->g = srcColor.g * (1 - t) + destColor.g * t;
    rgb->r = srcColor.r * (1 - t) + destColor.r * t;
    rgb->a = 255;
}

/** Parses all arguments and reads the input images and lines */
void parseAndReadFiles(int argc, char *argv[])
{
    printf("\n");
    if (!(argc == 6 || argc == 9)) // has to be either 6 or 9
    {
        cout << "Usage: ./morph source.png destination.png lines.txt outputPath steps [p] [a] [b]" << endl;
        exit(1);
    }
    string fileSourceImage = argv[1];
    string fileDestinationImage = argv[2];
    string fileLines = argv[3];
    outputPath = argv[4];
    istringstream(argv[5]) >> steps;
    stepSize = 1.0 / steps;
    if (argc == 9)
    {
        istringstream(argv[6]) >> p;
        istringstream(argv[7]) >> a;
        istringstream(argv[8]) >> b;
    }
    else
    {
        p = 0;
        a = 1;
        b = 2;
    }
    t = 0.5;
    imgRead(fileSourceImage, sourceImage, imageWidth, imageHeight);
    imgRead(fileDestinationImage, destinationImage, imageWidth, imageHeight);
    loadLines(fileLines.c_str(), sourceLines, destinationLines, &numLines);
}

__global__ void morphKernel(SimpleFeatureLine *sourceLines,
                            SimpleFeatureLine *destinationLines,
                            SimpleFeatureLine *morphLines,
                            pixel *sourceImage,
                            pixel *destinationImage,
                            pixel *morphedImage,
                            int imageWidth, int imageHeight,
                            int numLines, float dT)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    if (imageWidth <= x || imageHeight <= y)
        return; // Out of range of image

    extern __shared__ SimpleFeatureLine lines[]; // Shared memory for all feature lines
    // Split shared memory into three sections for the three feature lines
    SimpleFeatureLine *sSrcLines = &lines[0 * numLines];
    SimpleFeatureLine *sDstLines = &lines[1 * numLines];
    SimpleFeatureLine *sMrpLines = &lines[2 * numLines];

    // Each thread fills a specific index from the global to shared memory for its block
    // This assumes that there are more pixels/threads in each block than lines, which i would
    // say is a fair assumption (in Part 2 number of threads per block is 64, and numLines is 33)
    int blockIndex = threadIdx.y * blockDim.x + threadIdx.x;
    if (blockIndex < numLines)
    {
        sSrcLines[blockIndex] = sourceLines[blockIndex];
        sDstLines[blockIndex] = destinationLines[blockIndex];
        sMrpLines[blockIndex] = morphLines[blockIndex];
    }
    __syncthreads(); // wait for all shared memory to be ready in this block

    SimplePoint q{.x = float(x), .y = float(y)};
    SimplePoint src, dest;

    warp(&q, sMrpLines, sSrcLines, numLines, &src);
    warp(&q, sMrpLines, sDstLines, numLines, &dest);

    src.x = CLAMP<double>(src.x, 0, imageWidth - 1);
    src.y = CLAMP<double>(src.y, 0, imageHeight - 1);
    dest.x = CLAMP<double>(dest.x, 0, imageWidth - 1);
    dest.y = CLAMP<double>(dest.y, 0, imageHeight - 1);

    pixel interColor;
    ColorInterPolate(&src, &dest, dT, sourceImage, destinationImage, &interColor, imageWidth);

    int index = y * imageWidth + x;
    morphedImage[index].r = interColor.r;
    morphedImage[index].g = interColor.g;
    morphedImage[index].b = interColor.b;
    morphedImage[index].a = interColor.a;
}

void performMorphing(int argc, char *argv[])
{
    parseAndReadFiles(argc, argv);

    // Calculate all sizes
    size_t morphArrSize = sizeof(pixel *) * (steps + 1);
    size_t lineArrSize = sizeof(SimpleFeatureLine *) * (steps + 1);
    size_t imageSize = sizeof(pixel) * imageWidth * imageHeight;
    size_t lineSize = sizeof(SimpleFeatureLine) * numLines;
    // Shared memory will contain sourceLines, destinationLines and morphLines
    size_t sharedMemSize = 3 * lineSize;

    // Create arrays for all outputimages and all the morph lines
    morphedImages = (pixel **)malloc(morphArrSize);
    allMorphLines = (SimpleFeatureLine **)malloc(lineArrSize);
    for (int i = 0; i < steps + 1; i++)
    {
        morphedImages[i] = (pixel *)malloc(imageSize);
        simpleLineInterpolate(sourceLines, destinationLines, &(allMorphLines[i]), numLines, t);
    }

    // Allocate space on device (GPU) for lines and images
    pixel *dSourceImage, *dDestinationImage, *dMorphedImage;
    hipMalloc((void **)&dSourceImage, imageSize);
    hipMalloc((void **)&dDestinationImage, imageSize);
    hipMalloc((void **)&dMorphedImage, imageSize);
    SimpleFeatureLine *dSourceLines, *dDestinationLines, *dMorphLines;
    hipMalloc((void **)&dSourceLines, lineSize);
    hipMalloc((void **)&dDestinationLines, lineSize);
    hipMalloc((void **)&dMorphLines, lineSize);

    // Copy source and destination data to device
    hipMemcpy(dSourceImage, sourceImage, imageSize, hipMemcpyHostToDevice);
    hipMemcpy(dSourceLines, sourceLines, lineSize, hipMemcpyHostToDevice);
    hipMemcpy(dDestinationImage, destinationImage, imageSize, hipMemcpyHostToDevice);
    hipMemcpy(dDestinationLines, destinationLines, lineSize, hipMemcpyHostToDevice);

    // Defining Block and Grid Size
    dim3 blockSize(8, 8);
    int num_blocks_x = (imageWidth / blockSize.x);
    int num_blocks_y = (imageHeight / blockSize.y);
    dim3 gridSize(num_blocks_x, num_blocks_y);

    for (int i = 0; i < steps + 1; i++)
    {
        // Copy morph lines for this step to device
        hipMemcpy(dMorphLines, allMorphLines[i], lineSize, hipMemcpyHostToDevice);

        // Launching Kernel
        morphKernel<<<gridSize, blockSize, sharedMemSize>>>(
            dSourceLines, dDestinationLines, dMorphLines,
            dSourceImage, dDestinationImage, dMorphedImage,
            imageWidth, imageHeight, numLines, i * stepSize);

        // Copy morphed image for this step from device to host
        hipMemcpy(morphedImages[i], dMorphedImage, imageSize, hipMemcpyDeviceToHost);

        printProgress("Morphing Images", i + 1, steps + 1);
    }

    // Free all Cuda memory
    hipFree(dSourceImage);
    hipFree(dSourceLines);
    hipFree(dDestinationImage);
    hipFree(dDestinationLines);
    hipFree(dMorphedImage);
    hipFree(dMorphLines);
}



/////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////// ALL RELEVANT CODE IS BELOW /////////////////////////////////////////
//////////////// Above is only code from previous assignment ////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////



// Struct for arguments for pthread for writing image to file
typedef struct Thread_struct
{
    string filename;
    pixel *image;
} Thread;

// Track progress of threads using a shared counter
int numImagesWritten = 0;
pthread_mutex_t mutex;

/** Writes a single image to file */
void *pthread_write_image(void *args)
{
    // Deconstruct arguments
    Thread thread = *(Thread *)args;
    // Write to file 
    imgWrite(thread.filename, thread.image, imageWidth, imageHeight);

    // Update counter and print current progress
    pthread_mutex_lock(&mutex);
    {
        numImagesWritten += 1;
        printProgress("\tpthreads", numImagesWritten, steps + 1);
    }
    pthread_mutex_unlock(&mutex);
    return NULL;
}

/**
 * MAIN
 * 
 * performs the morphing then writes the morphed images to file, 
 * first serially, then using pthreads
 */
int main(int argc, char *argv[])
{
    performMorphing(argc, argv);
    printf("Writing To File:\n");

    // SERIAL //////////////////////////////////////////////////////////////////////
    struct timeval start, end;   
    gettimeofday(&start, NULL);
    {
        // Write the morphed images to file and free the host memory
        printProgress("\tSerial", 0, steps + 1);
        for (int i = 0; i < steps + 1; i++)
        {
            string filename = outputPath + to_string(stepSize * i) + ".png";
            imgWrite(filename, morphedImages[i], imageWidth, imageHeight);
            printProgress("\tSerial", i + 1, steps + 1);
        }
    }
    gettimeofday(&end, NULL);
    double serial_time = WALLTIME(end) - WALLTIME(start);
    printf("\tTime: \t%.2f seconds\n", serial_time);

        
    
    // PTHREADS ////////////////////////////////////////////////////////////////////
    gettimeofday(&start, NULL);
    {
        // Allocate space for threads and their argments
        pthread_t thread_id[steps + 1];
        Thread thread_args[steps + 1];

        printProgress("\tpthreads", 0, steps + 1);
        for (int i = 0; i < steps + 1; i++)
        {
            // Set the arguments
            thread_args[i].filename = outputPath + to_string(stepSize * i) + ".png";
            thread_args[i].image = morphedImages[i];
            // Create thread that writes an image to file
            pthread_create(&thread_id[i], NULL, &pthread_write_image, &thread_args[i]);
        }

        // Join the threads (wait until everyone is finished)
        for (int i = 0; i < steps + 1; i++) pthread_join(thread_id[i], NULL);
    }
    gettimeofday(&end, NULL);
    double pthread_time = WALLTIME(end) - WALLTIME(start);
    printf("\tTime: \t%.2f seconds \t(%.2f x Faster)\n", pthread_time, serial_time/pthread_time);



    // Free all memory
    for(int i = 0; i < steps + 1; i++)
    {
        free(morphedImages[i]);
        free(allMorphLines[i]);
    }
    free(morphedImages);
    free(allMorphLines);

    return 0;
}
