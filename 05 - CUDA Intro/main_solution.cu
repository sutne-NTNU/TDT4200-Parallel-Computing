#include "hip/hip_runtime.h"
#include <stdio.h>

#define STB_IMAGE_IMPLEMENTATION
#include "libs/stb/stb_image.h"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "libs/stb/stb_image_write.h"

typedef struct pixel_struct
{
    unsigned char r;
    unsigned char g;
    unsigned char b;
    unsigned char a;
} pixel;

#define cudaErrCheck(ans)                     \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

//////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////

__device__ void bilinear( // TODO 2 b: Change to device function
    pixel *image,
    float row,
    float col,
    pixel *pixel,
    int width,
    int height)
{
    int cm = (int)ceil(row);
    int fm = (int)floor(row);
    int cn = (int)ceil(col);
    int fn = (int)floor(col);
    double alpha = ceil(row) - row;
    double beta = ceil(col) - col;

    pixel->r = (unsigned char)(alpha * beta * image[fm * width + fn].r                 //
                               + (1 - alpha) * beta * image[cm * width + fn].r         //
                               + alpha * (1 - beta) * image[fm * width + cn].r         //
                               + (1 - alpha) * (1 - beta) * image[cm * width + cn].r); //
    pixel->g = (unsigned char)(alpha * beta * image[fm * width + fn].g                 //
                               + (1 - alpha) * beta * image[cm * width + fn].g         //
                               + alpha * (1 - beta) * image[fm * width + cn].g         //
                               + (1 - alpha) * (1 - beta) * image[cm * width + cn].g); //
    pixel->b = (unsigned char)(alpha * beta * image[fm * width + fn].b                 //
                               + (1 - alpha) * beta * image[cm * width + fn].b         //
                               + alpha * (1 - beta) * image[fm * width + cn].b         //
                               + (1 - alpha) * (1 - beta) * image[cm * width + cn].b); //
    pixel->a = 255;                                                                    //
}
/////////////////////////////////////////////////////////////////////////////////////////

__global__ void bilinear_kernel( // TODO 2 a: Change to kernel
    pixel *device_pixels_in,
    pixel *device_pixels_out,
    int in_width,
    int in_height,
    int out_width,
    int out_height)
{
    // TODO 2 c - Parallelize the kernel ////////////////////////////////////////////////
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    bool pixel_out_of_range = (out_width <= x) || (out_height <= y);
    if (pixel_out_of_range)
        return;

    pixel new_pixel;
    float col = x * (in_width - 1) / (float)out_width;
    float row = y * (in_height - 1) / (float)out_height;
    bilinear(device_pixels_in, row, col, &new_pixel, in_width, in_height);
    device_pixels_out[y * out_width + x] = new_pixel;
    /////////////////////////////////////////////////////////////////////////////////////
}

int main(int argc, char **argv)
{
    stbi_set_flip_vertically_on_load(true);
    stbi_flip_vertically_on_write(true);

    int in_width, in_height, channels;
    pixel *host_pixels_in;

    host_pixels_in = (pixel *)stbi_load(argv[1], &in_width, &in_height, &channels, STBI_rgb_alpha);
    if (host_pixels_in == NULL)
    {
        exit(1);
    }
    printf("Image dimensions: %dx%d\n", in_width, in_height);

    double scale_x = argc > 2 ? atof(argv[2]) : 2;
    double scale_y = argc > 3 ? atof(argv[3]) : 8;

    int out_width = in_width * scale_x;
    int out_height = in_height * scale_y;

    long size_in = sizeof(pixel) * in_width * in_height;
    long size_out = sizeof(pixel) * out_width * out_height;

    pixel *host_pixels_out = (pixel *)malloc(size_out);

    // TODO 1 a - cuda malloc ///////////////////////////////////////////////////////////
    pixel *device_pixels_in;
    pixel *device_pixels_out;
    hipMalloc((void **)&device_pixels_in, size_in);
    hipMalloc((void **)&device_pixels_out, size_out);
    ////////////////////////////////////////////////////////////////////////////////////

    hipEvent_t start_transfer, stop_transfer;
    hipEventCreate(&start_transfer);
    hipEventCreate(&stop_transfer);
    hipEventRecord(start_transfer);

    // TODO 1 b - cuda memcpy /////////////////////////////////////////////////////////
    hipMemcpy(device_pixels_in, host_pixels_in, size_in, hipMemcpyHostToDevice);
    ///////////////////////////////////////////////////////////////////////////////////

    // TODO 1 c - block size and grid size. ///////////////////////////////////////////
    // gridSize should depend on the blockSize and output dimensions.
    dim3 blockSize(32, 32);
    int num_blocks_x = (out_width / blockSize.x);
    int num_blocks_y = (out_height / blockSize.y);
    dim3 gridSize(num_blocks_x, num_blocks_y);
    ///////////////////////////////////////////////////////////////////////////////////

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // TODO 2 a - GPU computation /////////////////////////////////////////////////////
    // Change the function call so that it becomes a kernel call. Change the input
    // and output pixel variables to be device-side instead of host-side.
    bilinear_kernel<<<gridSize, blockSize>>>(device_pixels_in, device_pixels_out, in_width, in_height, out_width, out_height);
    ////////////////////////////////////////////////////////////////////////////////////

    hipEventRecord(stop);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        printf("%s\n", hipGetErrorString(err));
    hipDeviceSynchronize();
    hipEventSynchronize(stop);
    float spentTime = 0.0;
    hipEventElapsedTime(&spentTime, start, stop);
    printf("Time spent %.3f seconds\n", spentTime / 1000);

    // TODO 3 a - Copy the device-side data into the host-side variable ////////////////
    hipMemcpy(host_pixels_out, device_pixels_out, size_out, hipMemcpyDeviceToHost);
    ////////////////////////////////////////////////////////////////////////////////////

    hipEventRecord(stop_transfer);
    hipEventSynchronize(stop_transfer);
    float spentTimeTransfer = 0.0;
    hipEventElapsedTime(&spentTimeTransfer, start_transfer, stop_transfer);
    printf("Time spent including transfer: %.3f seconds\n", spentTimeTransfer / 1000);
    stbi_write_png("output.png", out_width, out_height, STBI_rgb_alpha, host_pixels_out, sizeof(pixel) * out_width);

    // TODO 3 b - Free heap-allocated memory on device and host ////////////////////////
    free(host_pixels_in);
    free(host_pixels_out);
    hipFree(device_pixels_in);
    hipFree(device_pixels_out);
    ////////////////////////////////////////////////////////////////////////////////////

    return 0;
}
