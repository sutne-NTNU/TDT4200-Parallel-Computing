#include "hip/hip_runtime.h"
#include <stdio.h>

#define STB_IMAGE_IMPLEMENTATION
#include "libs/stb/stb_image.h"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "libs/stb/stb_image_write.h"

typedef struct pixel_struct
{
    unsigned char r;
    unsigned char g;
    unsigned char b;
    unsigned char a;
} pixel;

void bilinear(pixel *Im, float row, float col, pixel *pix, int width, int height)
{
    int cm = (int)ceil(row);
    int fm = (int)floor(row);
    int cn = (int)ceil(col);
    int fn = (int)floor(col);
    double alpha = ceil(row) - row;
    double beta = ceil(col) - col;

    pix->r = (unsigned char)(alpha * beta * Im[fm * width + fn].r                 //
                             + (1 - alpha) * beta * Im[cm * width + fn].r         //
                             + alpha * (1 - beta) * Im[fm * width + cn].r         //
                             + (1 - alpha) * (1 - beta) * Im[cm * width + cn].r); //
    pix->g = (unsigned char)(alpha * beta * Im[fm * width + fn].g                 //
                             + (1 - alpha) * beta * Im[cm * width + fn].g         //
                             + alpha * (1 - beta) * Im[fm * width + cn].g         //
                             + (1 - alpha) * (1 - beta) * Im[cm * width + cn].g); //
    pix->b = (unsigned char)(alpha * beta * Im[fm * width + fn].b                 //
                             + (1 - alpha) * beta * Im[cm * width + fn].b         //
                             + alpha * (1 - beta) * Im[fm * width + cn].b         //
                             + (1 - alpha) * (1 - beta) * Im[cm * width + cn].b); //
    pix->a = 255;                                                                 //
}

void bilinear_kernel(pixel *d_pixels_in, pixel *d_pixels_out,
                     int in_width, int in_height,
                     int out_width, int out_height)
{
    for (int i = 0; i < out_height; i++)
    {
        for (int j = 0; j < out_width; j++)
        {
            pixel new_pixel;

            float row = i * (in_height - 1) / (float)out_height;
            float col = j * (in_width - 1) / (float)out_width;

            bilinear(d_pixels_in, row, col, &new_pixel, in_width, in_height);

            d_pixels_out[i * out_width + j] = new_pixel;
        }
    }
}

int main(int argc, char **argv)
{
    stbi_set_flip_vertically_on_load(true);
    stbi_flip_vertically_on_write(true);

    int in_width;
    int in_height;

    pixel *h_pixels_in;
    int channels;
    h_pixels_in = (pixel *)stbi_load(argv[1], &in_width, &in_height, &channels, STBI_rgb_alpha);
    if (h_pixels_in == NULL)
        exit(1);

    printf("Image dimensions: %dx%d\n", in_width, in_height);

    double scale_x = argc > 2 ? atof(argv[2]) : 1;
    double scale_y = argc > 3 ? atof(argv[3]) : 1;

    int out_width = in_width * scale_x;
    int out_height = in_height * scale_y;

    pixel *h_pixels_out = (pixel *)malloc(sizeof(pixel) * out_width * out_height);

    clock_t start = clock();
    bilinear_kernel(h_pixels_in, h_pixels_out, in_width, in_height, out_width, out_height);
    clock_t end = clock();

    float time = ((float)(end - start)) / CLOCKS_PER_SEC;
    printf("Time spent %.3f seconds\n", time);

    stbi_write_png("output.png", out_width, out_height, STBI_rgb_alpha, h_pixels_out, sizeof(pixel) * out_width);
    return 0;
}
